extern "C"


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

class DistanceData {
public:
	int key;
	float distance;
};

//extern "C" __device__ float fmod2(float x, float y) {
//	return x - trunc(x / y) * y;
//}
//
//extern "C" __device__ float rnd(int seed) {
//	int a = seed;
//	a = (a ^ 61) ^ (a >> 16);
//	a = a + (a << 3);
//	a = a ^ (a >> 4);
//	a = a * 0x27d4eb2d;
//	a = a ^ (a >> 15);
//
//	return (float) a / 2147483647;
//}
//
//extern "C" __device__ float getDaytimeLightFactor(float *worldData) {
//	return 1;
//}
//
//extern "C" __device__ float clamp(float min, float max, float f) {
//	if (f > max)
//		return max;
//	if (f < min)
//		return min;
//	return f;
//}
//
//extern "C" __device__ float getLightAt(float *worldData, float y) {
//	float minY = worldData[1];
//	float height = worldData[5];
//	float yValue = abs(y - minY) / (height);
//	yValue = clamp(0, 1, yValue);
//
//	float lightLevels = pow(yValue, 16);
//	lightLevels = clamp(0, 1, lightLevels);
//
//	return lightLevels * getDaytimeLightFactor(worldData);
//}
//
//extern "C" __device__ float getChemicalAt(float *worldData, float y) {
//	float maxY = worldData[3];
//	float height = worldData[5];
//	float yValue = abs(y - maxY) / (height);
//	yValue = clamp(0, 1, yValue);
//
//	float chemicalLevels = pow(yValue, 16);
//	chemicalLevels = clamp(0, 1, chemicalLevels);
//
//	return chemicalLevels;
//}
//
//extern "C" __device__ float relativeThetaAngleTo(float from, float to) {
//	// double angle = Math.atan2(to.y, to.x) - Math.atan2(from.y, from.x);
//	float angle = from - to;
//	float PI = 3.14159265359;
//
//	if (angle > PI)
//		angle -= 2 * PI;
//	else if (angle < -PI)
//		angle += 2 * PI;
//
//	return angle;
//}
//
//extern "C" __device__ float2 rotate(float2 center, float2 location,
//		float theta) {
//	float cs = cos(theta);
//	float sn = sin(theta);
//
//	float translated_x = location.x - center.x;
//	float translated_y = location.y - center.y;
//
//	float result_x = translated_x * cs - translated_y * sn;
//	float result_y = translated_x * sn + translated_y * cs;
//
//	result_x += center.x;
//	result_y += center.y;
//
//	return make_float2(result_x, result_y);
//}
//
//extern "C" __device__ float3 HSVtoRGB(float H, float S, float V) {
//	float s = S / 100;
//	float v = V / 100;
//	float C = s * v;
//	float X = C * (1 - abs(fmod2(H / 60.0, 2) - 1));
//	float m = v - C;
//	float r, g, b;
//	if (H >= 0 && H < 60) {
//		r = C, g = X, b = 0;
//	} else if (H >= 60 && H < 120) {
//		r = X, g = C, b = 0;
//	} else if (H >= 120 && H < 180) {
//		r = 0, g = C, b = X;
//	} else if (H >= 180 && H < 240) {
//		r = 0, g = X, b = C;
//	} else if (H >= 240 && H < 300) {
//		r = X, g = 0, b = C;
//	} else {
//		r = C, g = 0, b = X;
//	}
//
//	return make_float3(r + m, g + m, b + m);
//}
//
//extern "C" __device__ float relu(float value) {
//	if (value < 0)
//		return 0;
//	else
//		return value;
//}
//
//extern "C" __device__ void respawnAgent(int threadID, int startIndex,
//		float *inputs, float *networks, float *worldData) {
//	worldData[6]++; // decrease increment of living agents
//
//	// overwrite the agent at this index with our own info
//	float locX = inputs[startIndex + 0];
//	float locY = inputs[startIndex + 1];
//	float velX = inputs[startIndex + 2];
//	float velY = inputs[startIndex + 3];
//	float rwidth = inputs[startIndex + 4]; // width of hitbox after being rotated
//	float rheight = inputs[startIndex + 5]; // height of hitbox after being rotated
//	float lightBlocked = inputs[startIndex + 6]; // blocking light factor
//	float chemsBlocked = inputs[startIndex + 7]; // blocking chems factor
//	float age = inputs[startIndex + 8]; // age
//	float energy = inputs[startIndex + 9]; // energy
//	float rotation = inputs[startIndex + 10]; // rotation
//
//	// respawn random location
//	inputs[startIndex + 0] = rnd((int) (worldData[7] + threadID) + 1)
//			* worldData[4] + worldData[0];
//	inputs[startIndex + 1] = rnd((int) (worldData[7] + threadID) + 2)
//			* worldData[5] + worldData[1];
//
//	inputs[startIndex + 8] = 1; // age to 1
//	inputs[startIndex + 9] = 10; // energy
//	inputs[startIndex + 10] = rnd((int) (worldData[7] + threadID) + 3) * 3.14
//			* 2;
//
//	inputs[startIndex + 12] = 1;
//
//}
//
//extern "C" __device__ void calculateForAgent(int threadID, int startIndex,
//		int totalAgents, int lengthOfAgentData, float *inputs, float *neighbors,
//		float *worldData, float *networks) {
//
//	// get misc variables from array
//
//	float locX = inputs[startIndex + 0];
//	float locY = inputs[startIndex + 1];
//	float velX = inputs[startIndex + 2];
//	float velY = inputs[startIndex + 3];
//	float width = inputs[startIndex + 4]; // width of hitbox after being rotated
//	float height = inputs[startIndex + 5]; // height of hitbox after being rotated
//	float lightBlocked = inputs[startIndex + 6]; // blocking light factor
//	float chemsBlocked = inputs[startIndex + 7]; // blocking chems factor
//	float age = inputs[startIndex + 8]; // age
//	float energy = inputs[startIndex + 9]; // energy
//	float rotation = inputs[startIndex + 10]; // rotation
//	float sightRange = inputs[startIndex + 11]; // sight range
//	bool living = inputs[startIndex + 12] == 1;
//	float hue = inputs[startIndex + 13]; // hue
//	float currentMetabolism = inputs[startIndex + 14]; // metabolism
//	float baseMetabolism = inputs[startIndex + 15]; // base metabolism
//	int segmentCount = inputs[startIndex + 16];
//	float density = inputs[startIndex + 17];
//	int photoCells = inputs[startIndex + 18];
//	int chemoCells = inputs[startIndex + 19];
//	int huntingCells = inputs[startIndex + 20];
//	float dietGene = inputs[startIndex + 21];
//	float maturity = inputs[startIndex + 22];
//	int indexAfterData = 23;
//
//	float2 location = make_float2(locX, locY);
//	float2 velocity = make_float2(velX, velY);
//
//	/*
//	 *
//	 * Calculate inputs for neural network. They are stored at the end of all the other inputs, right before the outputs
//	 *
//	 */
//	int nearbyAgents = 0;
//	float closestDist = neighbors[threadID * (totalAgents * 2) + 1];
//	int key = -1;
//	for (int i = 0; i < totalAgents; i++) {
//		if (i == threadID)
//			continue; // don't include self
//		float d = neighbors[threadID * totalAgents + i];
//		if (d < closestDist) {
//			d = closestDist;
//			key = i * totalAgents;
//		}
//		if (d < sightRange)
//			nearbyAgents++;
//	}
//
//	int totalNeurons = networks[0];
//	int maxConnections = networks[1];
//	int inputNeurons = networks[2];
//	int outputNeurons = networks[3];
//
//	float *neuralInput = new float[inputNeurons];
//
//	int index = 0;
//	if (closestDist < sightRange && key != -1) {
//		float2 relClosest = make_float2(inputs[key + 0], inputs[key + 1]);
//		relClosest.x -= inputs[key + 0];
//		relClosest.y -= inputs[key + 1];
//		relClosest = rotate(make_float2(0, 0), relClosest, -rotation);
//
//		float3 color = HSVtoRGB(hue * 360, 100, 100);
//
//		neuralInput[0] = color.x;
//		neuralInput[1] = color.y;
//		neuralInput[2] = color.z;
//		neuralInput[3] = relClosest.x / sightRange;
//		neuralInput[4] = relClosest.y / sightRange;
//		neuralInput[5] = relativeThetaAngleTo(rotation, inputs[key + 10]); // show entity's direction relative to our own
//		neuralInput[6] = 1.0 / (1 + inputs[key + 9] / 10000.0); // agent energy
//		neuralInput[7] = inputs[key + 2] / sightRange; // velocityX / sightrange
//		neuralInput[8] = inputs[key + 3] / sightRange; // velocityY / sightrange
//	} else {
//		neuralInput[0] = 0; // r
//		neuralInput[1] = 0; // g
//		neuralInput[2] = 0; // b
//		neuralInput[3] = 0;
//		neuralInput[4] = 0;
//		neuralInput[5] = 0;
//		neuralInput[6] = 1;
//		neuralInput[7] = 1;
//		neuralInput[8] = 1;
//	}
//
//	neuralInput[9] = getLightAt(worldData, locY);
//	neuralInput[10] = getChemicalAt(worldData, locY);
//	neuralInput[11] = velocity.x;
//	neuralInput[12] = velocity.y;
//	neuralInput[13] = 1.0 / (1 + age);
//	neuralInput[14] = 1.0 / (1 + energy);
//	neuralInput[15] = 1.0 / (1 + nearbyAgents);
//	neuralInput[16] = relativeThetaAngleTo(3.14159265359 / 2, rotation); // to light is just their direction relative to up
//	neuralInput[17] = 0; //wall == null ? 1 : distToWall / (sightRange * sightRange);// world.getFoodNoise(getLocation().x, getLocation().y);
//	neuralInput[18] = 0; // direction of current at current spot
//	neuralInput[19] = 0; // direction of sun
//	neuralInput[20] = 1; // bias
//
//	/// [neuronID,output,function, connWeight,connFrom,connTo, connWeight,connFrom,connTo...]
//	// [0,1,0, .5,0,2, -1,-1,-1]
//
//	int lengthPerNeuron = 3 + maxConnections * 3;
//	int lengthPerNetwork = totalNeurons * lengthPerNeuron;
//
//	// calculate neural network
//	int networkBegin = 4 + threadID * lengthPerNetwork;
//
//	// put network inputs into input array
//	for (int i = 0; i < networks[2]; i++) {
////		inputs[startIndex + indexAfterData + i] = neuralInput[i];
//		networks[networkBegin + (i * lengthPerNeuron) + 1] = neuralInput[i];
//	}
//
//	for (int i = 0; i < totalNeurons; i++) {
//		int neuronID = networks[networkBegin + i * lengthPerNeuron + 0];
//		int neuronFunction = networks[networkBegin + i * lengthPerNeuron + 3];
//
//		float sum = 0;
//		// get every input connection for this neuron
//		for (int j = 0; j < maxConnections; j++) {
//			int to = networks[networkBegin + i * lengthPerNeuron
//					+ (3 + j * 3 + 2)]; // the neuron the connection is from
//			int from = networks[networkBegin + i * lengthPerNeuron
//					+ (3 + j * 3 + 1)]; // the neuron the connection is from
//			if (to != -1 && from != -1) {
//				float weight = networks[networkBegin + i * lengthPerNeuron
//						+ (3 + j * 3 + 0)]; // weight of connection
//				float out = networks[networkBegin + from * lengthPerNetwork + 1]; // output of from neuron
//				sum += weight;
//			}
//		}
//
//		float output;
//		if (neuronFunction == 0)
//			output = relu(sum);
//		else if (neuronFunction == 1)
//			output = tanh(sum);
//
//		networks[networkBegin + i * lengthPerNeuron + 1] = sum;
//	}
//
//	// get outputs from neurons
//	int outputIndex = 0;
//	for (int i = totalNeurons - outputNeurons; i < totalNeurons; i++) {
//		float out = networks[networkBegin + i * lengthPerNeuron + 1]; // out
////		inputs[startIndex + indexAfterData + inputNeurons + outputIndex] = out;
//		outputIndex++;
//	}
//
//	float fullMass = 8 * segmentCount * density;
//	maturity = ((age * 0.004) / fullMass);
//	float matureMass = 8 * segmentCount * density * (maturity + 0.001);
//	currentMetabolism = (0.5 + baseMetabolism)
//			* (1 - clamp(0, 1, maturity - 1));
//
//	// use brain outputs
//
//	float rotDesire = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 0) * lengthPerNeuron + 1];
//	float movementForce = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 1) * lengthPerNeuron + 1];
//
//	bool wantsToReproduce = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 2) * lengthPerNeuron + 1] >= 0.0;
//	bool wantsToDigest = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 3) * lengthPerNeuron + 1] >= 0;
//	bool wantsToEat = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 4) * lengthPerNeuron + 1] >= 0;
//	bool wantsToPhotosynthesize = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 5) * lengthPerNeuron + 1] >= 0;
//	bool wantsToChemosynthesize = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 6) * lengthPerNeuron + 1] >= 0;
//	bool wantsToEjectFood = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 7) * lengthPerNeuron + 1];
//	bool wantsToHeal = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 8) * lengthPerNeuron + 1];
//	bool wantsToAttack = networks[networkBegin
//			+ (totalNeurons - outputNeurons + 9) * lengthPerNeuron + 1] >= 0;
//
//	// update variables with brain
//
//	rotation += rotDesire * (3.14 * 0.01);
//
//	velX = cos(rotation) * movementForce * fullMass;
//	velY = sin(rotation) * movementForce * fullMass;
//
//	// tick
//	age++;
//
//	locX += velX;
//	locY += velY;
//	velX *= 1.0 * (1 / (1 + (segmentCount * 4)));
//	velY *= 1.0 * (1 / (1 + (segmentCount * 4)));
//
//	locX = clamp(worldData[0], worldData[2], locX);
//	locY = clamp(worldData[1], worldData[3], locY);
//
//	float sizeUsage = pow(matureMass * currentMetabolism * 0.03, 1.25);
//	float movementUsage = (movementForce * matureMass) * 0.0000003 * 0;
//	float used = (sizeUsage + movementUsage);
//
//	energy -= used;
//
//	float energyPerUnit = 15;
//	float competition = clamp(0, 1,
//			((float) segmentCount - nearbyAgents) / segmentCount);
//	float lightGain = photoCells * energyPerUnit * getLightAt(worldData, locY)
//			* competition;
//	float chemGain = chemoCells * energyPerUnit * getChemicalAt(worldData, locY)
//			* competition;
//
//	energy += lightGain + chemGain;
//
//	if (energy <= 0) {
//		living = false;
//	}
//
//	if (maturity > 2.0) {
//		living = false;
//	}
//
//	// set inputs
//
//	inputs[startIndex + 0] = locX;
//	inputs[startIndex + 1] = locY;
//	inputs[startIndex + 2] = velX;
//	inputs[startIndex + 3] = velY;
//	inputs[startIndex + 8] = age; // age
//	inputs[startIndex + 9] = energy; // energy
//	inputs[startIndex + 10] = rotation; // rotation
//	inputs[startIndex + 12] = living ? 1 : 0;
//	inputs[startIndex + 14] = currentMetabolism;
//	inputs[startIndex + 22] = maturity;
//
//	delete[] neuralInput;
//
//	if (!living) {
//		worldData[6]--; // decrease increment of living agents
//
//		// respawn if too low
//		respawnAgent(threadID, startIndex, inputs, networks, worldData);
//	}
//}
//
//extern "C" __global__ void tickAgents(float *inputs, int totalAgents,
//		int lengthOfAgentData, float *neighbors, float *worldData,
//		float *networks) {
//
//	int threadID = (blockIdx.x * blockDim.x + threadIdx.x);
//	if (threadID < totalAgents) {
//		int startIndex = threadID * lengthOfAgentData; // starting index for inputs
//		calculateForAgent(threadID, startIndex, totalAgents, lengthOfAgentData,
//				inputs, neighbors, worldData, networks);
//
//		worldData[7] += 1.0 / totalAgents;
//		return;
//	}
//
//}
//
//extern "C" __device__ void exchange(DistanceData *a, int i, int j) {
//	DistanceData t = a[i];
//	a[i] = a[j];
//	a[j] = t;
//}
//
//extern "C" __device__ void compare(DistanceData *a, int i, int j, bool dir) {
//	if (dir == (a[i].distance > a[j].distance))
//		exchange(a, i, j);
//}
//
//extern "C" __device__ void bitonicMerge(DistanceData *a, int lo, int n,
//		bool dir) {
//	if (n > 1) {
//		int m = n / 2;
//		for (int i = lo; i < lo + m; i++)
//			compare(a, i, i + m, dir);
//		bitonicMerge(a, lo, m, dir);
//		bitonicMerge(a, lo + m, m, dir);
//	}
//}
//
//extern "C" __device__ void bitonicSort(DistanceData *a, int lo, int n,
//		bool dir) {
//	if (n > lo) {
//		int m = (n - lo) / 2;
//		bitonicSort(a, lo, m, true);
//		bitonicSort(a, lo + m, m, false);
//		bitonicMerge(a, lo, n, dir);
//	}
//}
//
//extern "C" __device__ void sortDistances(DistanceData *a, int start, int end) {
//	bitonicSort(a, start, end, true);
//}

extern "C" __global__ void sortNeighbors(int totalAgents, float *neighbors,
		float *locations) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < totalAgents) {

		float array[10];
		for (int i = 0; i < 10; i++)
			array[i] = i;


		if (threadID == 0)
			printf("Length: %d %d %d\n", (sizeof(locations) / sizeof(float)), (sizeof(neighbors) / sizeof(float), (sizeof(array) / sizeof(float))));

	}

}

