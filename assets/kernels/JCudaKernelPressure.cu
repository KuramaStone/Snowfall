extern "C"


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

class DistanceData {
public:
	int key;
	float distance;
};

extern "C" __device__ float clamp2(float min, float max, float a) {
	if (a > max)
		return max;
	if (a < min)
		return min;
	return a;
}

extern "C" __device__ float fmod2(float x, float y) {
	return x - trunc(x / y) * y;
}

extern "C" __device__ float distance(float2 a, float2 b) {
	return pow(a.x - b.x, 2) + pow(a.y - b.y, 2);
}

extern "C" __global__ void addVelocity(int index, float x, float y,
		float *inputs) {
	inputs[index + 1] += x;
	inputs[index + 2] += y;
}

extern "C" __global__ void addDensity(int index, float amount, float *inputs) {
	inputs[index + 0] += amount;
}

extern "C" __global__ void update(int worldWidth, int worldHeight, int entries,
		float *inputs, float* outputs) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < (worldWidth * worldHeight)) {
		float myDensity = inputs[threadID * entries + 3]; // unaltered density
		// density of -1 indicates a wall
		if (myDensity == -1)
			return;

		// set last calculation's result as the new current last value
		inputs[threadID * entries + 3] = inputs[threadID * entries + 0];
		inputs[threadID * entries + 4] = inputs[threadID * entries + 1];
		inputs[threadID * entries + 5] = inputs[threadID * entries + 2];

		outputs[threadID] = clamp2(0, 1000000, inputs[threadID * entries + 0]);
	}
}

extern "C" __global__ void move(int worldWidth, int worldHeight, int entries,
		float *inputs) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < (worldWidth * worldHeight)) {
		float myDensity = inputs[threadID * entries + 3]; // unaltered density
		// density of -1 indicates a wall
		if (myDensity == -1)
			return;

		int x = (int) fmod2(threadID, worldWidth);
		int y = (int) (threadID / worldWidth);
		float vx = inputs[threadID * entries + 1];
		float vy = inputs[threadID * entries + 2];

		if (vx == 0 && vy == 0)
			return;

		// add density and velocity to (x+vx, y+vy)
		int x2 = x + vx;
		int y2 = y + vy;
		x2 = clamp2(0, worldWidth-1, x2);
		y2 = clamp2(0, worldHeight-1, y2);
		if (x2 == 0) {
			if (vx < 0)
				vx = 0;
		}
		if (x2 == worldWidth - 1) {
			if (vx > 0)
				vx = 0;
		}
		if (y2 == 0) {
			if (vy < 0)
				vy = 0;
		}
		if (y2 == worldHeight - 1) {
			if (vy > 0)
				vy = 0;
		}

		if(x2 == x && y2 == y)
			return;

		int id2 = (x2 + y2 * worldWidth) * entries;

		if (inputs[id2] == -1)
			return; // don't move into walls

		// my spot
		inputs[threadID * entries + 0] = myDensity / 2;

		// next spot
		inputs[id2 + 0] += myDensity / 2; // next density
		inputs[id2 + 1] += vx * 1; // next vx
		inputs[id2 + 2] += vy * 1; // next vy
	}
}

extern "C" __global__ void diffuse(int worldWidth, int worldHeight, int entries,
		float *inputs, float *output) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < (worldWidth * worldHeight)) {
		float myDensity = inputs[threadID * entries + 3]; // unaltered density
		// density of -1 indicates a wall
		if (myDensity == -1)
			return;

		int x = (int) fmod2(threadID, worldWidth);
		int y = (int) (threadID / worldWidth);

		float newDensity = myDensity;

		// diffuse. average nearby cells that aren't barriers
		float a = -1;
		if (x + 1 >= 0 && x + 1 < worldWidth) {
			a = inputs[(x + 1 + y * worldWidth) * entries + 3];
		}
		float b = -1;
		if (x - 1 >= 0 && x - 1 < worldWidth) {
			b = inputs[(x - 1 + y * worldWidth) * entries + 3];
		}
		float c = -1;
		if (y + 1 >= 0 && y + 1 < worldHeight) {
			c = inputs[(x + (y + 1) * worldWidth) * entries + 3];
		}
		float d = -1;
		if (y - 1 >= 0 && y - 1 < worldHeight) {
			d = inputs[(x + (y - 1) * worldWidth) * entries + 3];
		}

		int total = 5;
		if (a == -1) {
			total--;
			a = 0;
		}
		if (b == -1) {
			total--;
			b = 0;
		}
		if (c == -1) {
			total--;
			c = 0;
		}
		if (d == -1) {
			total--;
			d = 0;
		}

		newDensity = (a + b + c + d + myDensity) / total; // diffused amount

		inputs[threadID * entries + 0] = newDensity; // current density
	}

}

