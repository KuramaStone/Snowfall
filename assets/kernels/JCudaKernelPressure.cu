#include "hip/hip_runtime.h"
extern "C"

#include <math.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define NEARCOUNT 32

class DistanceData {
public:
	int key;
	float distance;
};

extern "C" __device__ int HSBtoRGB(float hue, float saturation,
		float brightness) {
	int r = 0, g = 0, b = 0;
	if (saturation == 0) {
		r = g = b = (int) (brightness * 255.0f + 0.5f);
	} else {
		float h = (hue - (float) floor(hue)) * 6.0f;
		float f = h - (float) floor(h);
		float p = brightness * (1.0f - saturation);
		float q = brightness * (1.0f - saturation * f);
		float t = brightness * (1.0f - (saturation * (1.0f - f)));
		switch ((int) h) {
		case 0:
			r = (int) (brightness * 255.0f + 0.5f);
			g = (int) (t * 255.0f + 0.5f);
			b = (int) (p * 255.0f + 0.5f);
			break;
		case 1:
			r = (int) (q * 255.0f + 0.5f);
			g = (int) (brightness * 255.0f + 0.5f);
			b = (int) (p * 255.0f + 0.5f);
			break;
		case 2:
			r = (int) (p * 255.0f + 0.5f);
			g = (int) (brightness * 255.0f + 0.5f);
			b = (int) (t * 255.0f + 0.5f);
			break;
		case 3:
			r = (int) (p * 255.0f + 0.5f);
			g = (int) (q * 255.0f + 0.5f);
			b = (int) (brightness * 255.0f + 0.5f);
			break;
		case 4:
			r = (int) (t * 255.0f + 0.5f);
			g = (int) (p * 255.0f + 0.5f);
			b = (int) (brightness * 255.0f + 0.5f);
			break;
		case 5:
			r = (int) (brightness * 255.0f + 0.5f);
			g = (int) (p * 255.0f + 0.5f);
			b = (int) (q * 255.0f + 0.5f);
			break;
		}
	}
	return 0xff000000 | (r << 16) | (g << 8) | (b << 0);
}

extern "C" __device__ float atan2f(float y, float x) {
	float a = 0;

	if (x > 0) {
		a = atan(y / x);
	} else if (x < 0 && y >= 0) {
		a = atan(y / x) + 3.14159265359;
	} else if (x < 0 && y < 0) {
		a = atan(y / x) - 3.14159265359;
	} else if (x == 0 && y > 0) {
		a = 3.14159265359 / 2;
	} else if (x == 0 && y < 0) {
		a = -3.14159265359 / 2;
	}

	return a;
}

extern "C" __device__ float clamp2(float min, float max, float a) {
	if (a > max)
		return max;
	if (a < min)
		return min;
	return a;
}

extern "C" __device__ float fmod2(float x, float y) {
	return x - trunc(x / y) * y;
}

extern "C" __device__ float distance(float2 a, float2 b) {
	return pow(a.x - b.x, 2) + pow(a.y - b.y, 2);
}

extern "C" __device__ int IX(int x, int y, int width, int entries) {
	return (x + y * width) * entries;
}

extern "C" __device__ float lerp(float i, float j, float a) {
	return i + a * (j - i);
}

extern "C" __device__ float min2(float i, float j) {
	if (i > j)
		return j;

	return i;
}
extern "C" __device__ void exchange(DistanceData *a, int i, int j) {
	DistanceData t = a[i];
	a[i] = a[j];
	a[j] = t;
}

extern "C" __device__ void compare(DistanceData *a, int i, int j, bool dir) {
	if (dir == (a[i].distance > a[j].distance))
		exchange(a, i, j);
}

extern "C" __device__ void bitonicMerge(DistanceData *a, int lo, int n,
		bool dir) {
	if (n > 1) {
		int m = n / 2;
		for (int i = lo; i < lo + m; i++)
			compare(a, i, i + m, dir);
		bitonicMerge(a, lo, m, dir);
		bitonicMerge(a, lo + m, m, dir);
	}
}

extern "C" __device__ void bitonicSort(DistanceData *a, int lo, int n,
		bool dir) {
	if (n > lo) {
		int m = (n - lo) / 2;
		bitonicSort(a, lo, m, true);
		bitonicSort(a, lo + m, m, false);
		bitonicMerge(a, lo, n, dir);
	}
}

extern "C" __device__ void sortDistances(DistanceData *a, int start, int end) {
	bitonicSort(a, start, end, true);
}

extern "C" __device__ float2 rotate(float2 center, float2 loc, float theta) {
	double cs = cos(theta);
	double sn = sin(theta);

	double translated_x = loc.x - center.x;
	double translated_y = loc.y - center.y;

	double result_x = translated_x * cs - translated_y * sn;
	double result_y = translated_x * sn + translated_y * cs;

	result_x += center.x;
	result_y += center.y;

	return make_float2(result_x, result_y);
}

extern "C" __global__ void sortNeighbors(int worldWidth, int worldHeight,
		int *worldData, int maxParticles, float *inputs, int entries,
		int *neighbors) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < worldData[0]) {
		float x = inputs[threadID * entries + 0];
		float y = inputs[threadID * entries + 1];
		float vx = inputs[threadID * entries + 2];
		float vy = inputs[threadID * entries + 3];
		float radius = inputs[threadID * entries + 4];
		float density = inputs[threadID * entries + 5];
		float mass = (radius * radius * 3.14159265359) * density;

		// really bad algorithm for now
		float maxRange = radius + radius * 20;
		float longRange2 = maxRange * maxRange;

		int nearbyIndex = 0;

		for (int i = 0; i < worldData[0]; i++) {
			int nearID = i;

			float x2 = inputs[nearID * entries + 0];
			float y2 = inputs[nearID * entries + 1];
			float r2 = inputs[nearID * entries + 4];

			float distRaw = pow(x - x2, 2) + pow(y - y2, 2);
			if (distRaw <= longRange2) {
				neighbors[threadID * NEARCOUNT + nearbyIndex] = nearID;

				if (nearbyIndex++ == NEARCOUNT)
					break;
			}
		}

	}
}

extern "C" __device__ float isOOB(int width, int height, float x, float y,
		float radius) {
	if (x < radius)
		return 6.28 / 4 * 2;
	if (x >= width - radius)
		return 6.28 / 4 * 0;
	if (y < radius)
		return 6.28 / 4 * 3;
	if (y >= height - radius)
		return 6.28 / 4 * 1;
	return -1;
}

extern "C" __global__ void applyForces(int worldWidth, int worldHeight,
		int *worldData, int maxParticles, float *inputs, int entries,
		int *neighbors) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < worldData[0]) {
		float x = inputs[threadID * entries + 0];
		float y = inputs[threadID * entries + 1];
		if (x == -1)
			return;
		float vx = inputs[threadID * entries + 2];
		float vy = inputs[threadID * entries + 3];
		float radius = inputs[threadID * entries + 4];

		// add new forces
		float vx2 = vx + inputs[threadID * entries + 6];
		float vy2 = vy + inputs[threadID * entries + 7];
//		vy2 += .2; // gravity

		// check if  it would take them out of bounds
		float collAngle = isOOB(worldWidth, worldHeight, x + vx2, y + vy2,
				radius);
		if (collAngle != -1) { // collides

			// rotate vx2, vy2 by collAngle and set vx2 to 0
			float2 result = rotate(make_float2(0, 0), make_float2(vx2, vy2),
					-collAngle);
			result.x = 0;
			result = rotate(make_float2(0, 0), result, collAngle); // rotate back and get final result
			vx2 = result.x;
			vy2 = result.y;
			vx2 = 0;
			vy2 = 0;

			// vx2 and vy2 will both no longer collide with the wall
		}
		x += vx2;
		y += vy2;

		float vel = pow(vx2, 2) + pow(vy2, 2);
		float resististance = 1.0 / (1 + pow(vel * 4, 2)); // slow down more as speed increases

		inputs[threadID * entries + 0] = x;
		inputs[threadID * entries + 1] = y;
		inputs[threadID * entries + 2] = vx2 * resististance;
		inputs[threadID * entries + 3] = vy2 * resististance;
		inputs[threadID * entries + 6] = 0;
		inputs[threadID * entries + 7] = 0;

	}
}

extern "C" __global__ void calculateForces(int worldWidth, int worldHeight,
		int *worldData, int maxParticles, float *inputs, int entries,
		int *neighbors) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < worldData[0]) {
		float x = inputs[threadID * entries + 0];
		float y = inputs[threadID * entries + 1];
		if (x == -1)
			return;
		float vx = inputs[threadID * entries + 2];
		float vy = inputs[threadID * entries + 3];
		float radius = inputs[threadID * entries + 4];
		float density = inputs[threadID * entries + 5];
		float mass = (radius * radius * 3.14159265359) * density;

		// calculate forces to other particles

		for (int i = 0; i < NEARCOUNT; i++) {
			int nearID = neighbors[threadID * NEARCOUNT + i];

			if (nearID != -1) {
				float x2 = inputs[nearID * entries + 0];
				float y2 = inputs[nearID * entries + 1];
				float vx2 = inputs[nearID * entries + 2];
				float vy2 = inputs[nearID * entries + 3];
				float r2 = inputs[nearID * entries + 4];
				float d2 = inputs[nearID * entries + 5];

				float distRaw = pow(x - x2, 2) + pow(y - y2, 2);

				float shortRangeRepulsionMax = radius + radius * 4 - r2;
				float force = 0.1;

				// if distance less than shortRange, pull it. Otherwise, it must be long range for it to be here
				if (distRaw <= pow(shortRangeRepulsionMax, 2)) {
					force = force / (1 + distRaw);

				} else {
					force = -force * distRaw / 1000;
				}

				float theta = atan2f(y - y2, x - x2);
				// get relative forces
				float fX = cos(theta) * force;
				float fY = sin(theta) * force;

				inputs[nearID * entries + 6] -= fX; // force to add next frame
				inputs[nearID * entries + 7] -= fY; // force to add next frame

			}

		}

	}

}

extern "C" __global__ void render(int worldWidth, int worldHeight,
		int *worldData, int maxParticles, float *inputs, int entries,
		char *output) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < worldWidth * worldHeight) {
		int x = fmod2(threadID, worldWidth);
		int y = threadID / worldWidth;

		// get distance to closest particle

		float heatSum = 0;

		for (int i = 0; i < worldData[0]; i++) {

			int id = i;

			if (id != -1) {
				float x2 = inputs[id * entries + 0];
				float y2 = inputs[id * entries + 1];
				float hue = inputs[id * entries + 8];

				float dist = pow(x - x2, 2) + pow(y - y2, 2);

				heatSum += 1 / (1 + dist);
			}
		}
		heatSum = clamp2(0.8, 1, heatSum);

		output[threadID] = (255 * heatSum) - 128;

	}
}

