extern "C"


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

class DistanceData {
public:
	int key;
	float distance;
};

extern "C" __device__ float fmod2(float x, float y) {
	return x - trunc(x / y) * y;
}

extern "C" __device__ float distance(float2 a, float2 b) {
	return pow(a.x - b.x, 2) + pow(a.y - b.y, 2);
}

extern "C" __global__ void calculate(float *inputs, int totalAgents,
		int entries, int hormones, int size, float *output) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < (size*size)) {
		float2 pixel = make_float2(fmod2(threadID, size), threadID / size);

		for (int i = 0; i < totalAgents; i++) {
			float x = inputs[i * entries + 0];
			float y = inputs[i * entries + 1];
			double dist = distance(pixel, make_float2(x, y));

			for (int j = 0; j < hormones; j++) {
				// get hormone level from distance
				double level = inputs[i * entries + 2 + j] / (1 + dist);

				output[threadID * hormones + j] += level;
			}

		}

	}

}

